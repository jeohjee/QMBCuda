#include "hip/hip_runtime.h"
#include "Heisenberg.h"
#include "thrust/complex.h"
#include "../quantum_operators/ArbitraryOperator.h"
#include <algorithm> 
#include <cmath>

using complex_th = thrust::complex<float>;

template <typename T> Heisenberg<T>::Heisenberg(LatticeGeometryInfo _geom_info, HeisenbergInfo<T> _heisenberg_info)
{
	geom_info = _geom_info;
	CreateGeometry();
	ComputeR_diffs();

	heisenberg_info = _heisenberg_info;
	CreateHamiltonian();
}
template Heisenberg<float>::Heisenberg(LatticeGeometryInfo _geom_info, HeisenbergInfo<float> _heisenberg_info);
template Heisenberg<complex_th>::Heisenberg(LatticeGeometryInfo _geom_info, HeisenbergInfo<complex_th> _heisenberg_info);


template <typename T>
void Heisenberg<T>::CreateHamiltonian() 
{
	// This function implements the creation of a generic XXZ Heisenberg lattice Hamiltonian.

	int M = geom_info.rx_alpha.size(); // number of sublattices
	int N1 = geom_info.N1;
	int N2 = geom_info.N2;
	int N = N1 * N2; // number of unit cells
	int LS = N1 * N2 * M; // number of sites

	bool intra_c_bool = heisenberg_info.intra_c_bool;
	bool inter_c_bool = heisenberg_info.inter_c_bool;

	if ((intra_c_bool == 0) && (inter_c_bool == 0)) {
		printf("ERROR: BOTH INTER AND INTRA-CLUSTER BOOLEANS ZERO, ABORTING");
		return;
	}

	std::vector<std::vector<std::vector<int>>> J_terms = { heisenberg_info.Jxy_terms, heisenberg_info.Jz_terms };

	std::vector<T> Jxy_couplings = heisenberg_info.Jxy_couplings;
	std::vector<float> Jz_couplings = heisenberg_info.Jz_couplings;

	for (int uc = 0; uc < N; uc++) {
		int j_site_1 = uc / N2;
		int j_site_2 = uc % N2;

		for (int jii = 0; jii < 2; jii++) {
			for (int ji = 0; ji < J_terms[jii].size(); ji++) {
				int j_site = uc * M + J_terms[jii][ji][3];

				int i_site_1 = j_site_1 + J_terms[jii][ji][0];
				int i_site_2 = j_site_2 + J_terms[jii][ji][1];
				if (!intra_c_bool && ( (i_site_1 < 0 || i_site_1 >= N1) || (i_site_2 < 0 || i_site_2 >= N2) )) continue;
				if (!inter_c_bool && ((i_site_1 >= 0 || i_site_1 < N1) || (i_site_2 >= 0 || i_site_2 < N2) )) continue;

				i_site_1 = ((i_site_1 % N1) + N1) % N1;
				i_site_2 = ((i_site_2 % N2) + N2) % N2;

				int i_site = J_terms[jii][ji][2] + M * (N2 * i_site_1 + i_site_2);


				if (jii == 0) {
					Sp<T> Sp_i = Sp<T>(i_site);
					Sm<T> Sm_j = Sm<T>(j_site);
					H = H + Jxy_couplings[ji] * (Sp_i * Sm_j);
				}
				else {
					Sz<T> Sz_i = Sz<T>(i_site);
					Sz<T> Sz_j = Sz<T>(j_site);
					H = H + Jz_couplings[ji] * (Sz_i * Sz_j);
				}
			}
		}
	}

}
template void Heisenberg<float>::CreateHamiltonian();
template void Heisenberg<complex_th>::CreateHamiltonian();


bool distance_comparator(const std::pair<int, float>& a, const std::pair<int, float>& b) {
	return a.second < b.second;
}

template <typename T>
Heisenberg<T> CreateExtendedHeisenbergXXX(LatticeGeometryInfo geom_info, std::vector<float> J_vec, bool intra_c_bool, bool inter_c_bool) {
	/*
	* This function creates Heisenberg<float> instance for the convetional extended XXX Heisenberg model in a lattice geometry determined by geom_info.
	* Args:
	*	N1, N2: number of unit cells in the directions of basis vectors
	*	J_vec: contains the spin-spin coupling strengths in the descending order. First element is the NN coupling, the
	*	next one is NNN coupling and so forth.
	*	intra_c_bool: determines whether the spin-spin couplings within the bulk are used or ignored (in most cases this should be true)
	*	inter_c_bool: determines whether periodic boundary conditions are used (true) or not (false).
	* This function should be easily generalizable for arbitrary lattices.
	*/

	std::vector<std::vector<float>> A_mat = geom_info.A_mat;
	HeisenbergInfo<T> heisenberg_info;
	int N1 = geom_info.N1;
	int N2 = geom_info.N2;
	int M = geom_info.rx_alpha.size();

	std::vector<std::vector<int>> all_coupling_terms;
	std::vector<int> i_vec;
	std::vector<int> j_vec;
	std::vector<int> alpha_vec;
	std::vector<int> beta_vec;
	std::vector<float> dist_vec;
	std::vector<int> indices;
	int curr_ind = 0;
	// CREATE POSSIBLE TERMS:

	for (int ni = -N1; ni < N1; ni++) {
		for (int nj = -N2; nj < N2; nj++) {

			for (int ai = 0; ai < M; ai++) {
				for (int bi = 0; bi < M; bi++) {
					all_coupling_terms.push_back({ni,nj,ai,bi});

					float tmp_dist = sqrtf(pow((float)ni * A_mat[0][0] + (float)nj * A_mat[0][1] + geom_info.rx_alpha[ai] - geom_info.rx_alpha[bi], 2)
						+ pow((float)ni * A_mat[1][0] + (float)nj * A_mat[1][1] + geom_info.ry_alpha[ai] - geom_info.ry_alpha[bi], 2));
					dist_vec.push_back(tmp_dist);
					indices.push_back(curr_ind);
					curr_ind = curr_ind + 1;
				}
			}

		}
	}
	// SORTING:
	int dist_size = dist_vec.size();
	// To sort i_vec, j_vec and dist_vec, we need to use the distance_comparator fuction:
	std::vector<std::pair<int, float>> distance_pairing;
	for (int ii = 0; ii < dist_size; ii++) {
		distance_pairing.emplace_back(indices[ii], dist_vec[ii]);
	}
	std::stable_sort(distance_pairing.begin(), distance_pairing.end(), distance_comparator);
	for (int ii = 0; ii < distance_pairing.size(); ii++) {
		indices[ii] = distance_pairing[ii].first; // needed to build final i_vec, j_vec, alpha_vec, beta_vec
		dist_vec[ii] = distance_pairing[ii].second;
	}
	std::vector<std::vector<int>> all_coupling_terms_copy = all_coupling_terms;
	for (int ii = 0; ii < distance_pairing.size(); ii++) {
		all_coupling_terms[ii] = all_coupling_terms_copy[indices[ii]];
	}

	// CHECK WHICH COUPLING TERMS ARE FEASIBLE:
	float J_curr = J_vec[0];
	int JSize = J_vec.size();
	int J_counter = -1;
	float dist_curr = dist_vec[0];
	float dist_thold = 0.001;
	// Over-complicated way to create the heisenberg_info struct:
	for (int i = 0; i < dist_size; i++) {
		if (dist_vec[i] < dist_thold) continue;
		if (dist_vec[i] > dist_curr + dist_thold) {
			J_counter = J_counter + 1;
			if (J_counter >= JSize) break;
			J_curr = J_vec[J_counter];

			dist_curr = dist_vec[i];
		}
		heisenberg_info.Jxy_terms.push_back(all_coupling_terms[i]);
		heisenberg_info.Jz_terms.push_back(all_coupling_terms[i]);
		heisenberg_info.Jxy_couplings.push_back((T)J_curr);
		heisenberg_info.Jz_couplings.push_back(J_curr);
	}
	heisenberg_info.intra_c_bool = intra_c_bool;
	heisenberg_info.inter_c_bool = inter_c_bool;
	return Heisenberg<T>(geom_info, heisenberg_info);

}
template Heisenberg<float> CreateExtendedHeisenbergXXX(LatticeGeometryInfo geom_info, std::vector<float> J_vec, bool intra_c_bool, bool inter_c_bool);
template Heisenberg<complex_th> CreateExtendedHeisenbergXXX(LatticeGeometryInfo geom_info, std::vector<float> J_vec, bool intra_c_bool, bool inter_c_bool);

template <typename T>
Heisenberg<T> CreateHeisenbergXXXSquare(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool, bool inter_c_bool)
{
	/*
	* This function creates Heisenberg<float> instance for a extended square lattice XXX Heisenberg model. 
	* Args:
	*	N1, N2: lattice size in the directions of the basis vectors.
	*	J_vec: contains the spin-spin coupling strengths in the descending order. First element is the NN coupling, the
	*	next one is NNN coupling and so forth.
	*	intra_c_bool: determines whether the spin-spin couplings within the bulk are used or ignored (in most cases this should be true)
	*	inter_c_bool: determines whether periodic boundary conditions are used (true) or not (false).
	*  
	*/
	LatticeGeometryInfo geom_info = create_square_lattice_info(N1, N2);
	return CreateExtendedHeisenbergXXX<T>(geom_info, J_vec, intra_c_bool, inter_c_bool);
}
template Heisenberg<float> CreateHeisenbergXXXSquare(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool=true, bool inter_c_bool=true);
template Heisenberg<complex_th> CreateHeisenbergXXXSquare(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool = true, bool inter_c_bool = true);

template <typename T>
Heisenberg<T> CreateHeisenbergXXXTriangular(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool, bool inter_c_bool)
{
	/*
	* This function creates Heisenberg<float> instance for a extended triangular lattice XXX Heisenberg model.
	* Args:
	*	N1, N2: lattice size in the directions of the basis vectors.
	*	J_vec: contains the spin-spin coupling strengths in the descending order. First element is the NN coupling, the
	*	next one is NNN coupling and so forth.
	*	intra_c_bool: determines whether the spin-spin couplings within the bulk are used or ignored (in most cases this should be true)
	*	inter_c_bool: determines whether periodic boundary conditions are used (true) or not (false).
	*
	*/
	LatticeGeometryInfo geom_info = create_triangular_lattice_info(N1, N2);
	return CreateExtendedHeisenbergXXX<T>(geom_info, J_vec, intra_c_bool, inter_c_bool);
}
template Heisenberg<float> CreateHeisenbergXXXTriangular(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool = true, bool inter_c_bool = true);
template Heisenberg<complex_th> CreateHeisenbergXXXTriangular(int N1, int N2, std::vector<float> J_vec, bool intra_c_bool = true, bool inter_c_bool = true);
