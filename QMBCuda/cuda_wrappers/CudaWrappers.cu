#include "CudaWrappers.h"
#include "thrust/complex.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../cublas_utils.h"
#include "../cusolver_utils.h"
#include <hipsolver.h>

using complex_th = thrust::complex<float>;

float VecNorm_cuBLAS_float_wrapper(float* __restrict d_A, int N) {

    hipblasHandle_t cublasH = NULL;
    //hipStream_t stream = NULL;

    const int incx = 1;
    float result = 0.0;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    //CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    //CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasSnrm2(cublasH, N, d_A, incx, &result));

    //CUDA_CHECK(hipStreamSynchronize(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    //CUDA_CHECK(hipStreamDestroy(stream));

    return result;
}

double VecNorm_cuBLAS_double_wrapper(double* __restrict d_A, int N) {

    hipblasHandle_t cublasH = NULL;
    //hipStream_t stream = NULL;

    const int incx = 1;
    double result = 0.0;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    //CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    //CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasDnrm2(cublasH, N, d_A, incx, &result));

    //CUDA_CHECK(hipStreamSynchronize(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    //CUDA_CHECK(hipStreamDestroy(stream));

    return result;
}

float VecNorm_cuBLAS_complex_float_wrapper(hipFloatComplex* __restrict d_A, int N) {

    hipblasHandle_t cublasH = NULL;

    const int incx = 1;
    float result = 0.0;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    //CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    //CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasScnrm2(cublasH, N, d_A, incx, &result));

    //CUDA_CHECK(hipStreamSynchronize(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    //CUDA_CHECK(hipStreamDestroy(stream));

    return result;
}


template <typename T>
float VecNorm_cuBLAS_generic_float_wrapper(T* __restrict d_A, int N) {

    float result = 0.0;
    if constexpr (::cuda::std::is_same_v<T, float>) {
        result = VecNorm_cuBLAS_float_wrapper(d_A, N);
        return result;
    }
    else {
        result = VecNorm_cuBLAS_complex_float_wrapper(reinterpret_cast<hipFloatComplex*>(d_A), N);
        return result;
    }

}
template float VecNorm_cuBLAS_generic_float_wrapper<float>(float* __restrict d_A, int N);
template float VecNorm_cuBLAS_generic_float_wrapper<complex_th>(complex_th* __restrict d_A, int N);

// Dot products
float DotProd_cuBLAS_float_wrapper(float* __restrict d_A, float* __restrict d_B, int N) {

    hipblasHandle_t cublasH = NULL;

    const int incx = 1;
    const int incy = 1;

    float result = 0.0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUBLAS_CHECK(hipblasSdot(cublasH, N, d_A, incx, d_B, incy, &result));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    return result;
}

double DotProd_cuBLAS_double_wrapper(double* __restrict d_A, double* __restrict d_B, int N) {

    hipblasHandle_t cublasH = NULL;

    const int incx = 1;
    const int incy = 1;

    double result = 0.0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUBLAS_CHECK(hipblasDdot(cublasH, N, d_A, incx, d_B, incy, &result));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    return result;
}

hipFloatComplex DotProd_cuBLAS_complex_wrapper(hipFloatComplex* __restrict d_A, hipFloatComplex* __restrict d_B, int N) {

    hipblasHandle_t cublasH = NULL;

    const int incx = 1;
    const int incy = 1;

    hipFloatComplex result;
    result.x = 0.0;
    result.y = 0.0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUBLAS_CHECK(hipblasCdotc(cublasH, N, d_A, incx, d_B, incy, &result));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    return result;
}

template <typename T>
T DotProd_cuBLAS_generic_float_wrapper(T* __restrict d_A, T* __restrict d_B, int N) {

    if constexpr (::cuda::std::is_same_v<T, float>) {
        float result;
        result = DotProd_cuBLAS_float_wrapper(d_A, d_B, N);
        return result;
    }
    else {
        hipFloatComplex result;
        result = DotProd_cuBLAS_complex_wrapper(
            reinterpret_cast<hipFloatComplex*>(d_A),
            reinterpret_cast<hipFloatComplex*>(d_B),
            N);
        return complex_th(result.x, result.y);
    }

}
template float DotProd_cuBLAS_generic_float_wrapper(float* __restrict d_A, float* __restrict d_B, int N);
template complex_th DotProd_cuBLAS_generic_float_wrapper(complex_th* __restrict d_A, complex_th* __restrict d_B, int N);

// Dense eigensolver for symmetric matrix:

template <typename T> 
void SymmDiag_cuSOLVER_wrapper(T* __restrict H_mat, T* __restrict W, T* __restrict V, const int N) {

    /* Parameters:
    * H_mat: a NxN matrix to be diagonalized
    * W: a vector of size N that will store the eigenvalues
    * V: NxN matrix for storing the eigenvectors
    */

    const int m = N;
    const int lda = N;

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;


    T* d_A = nullptr;
    T* d_W = nullptr;
    int* d_info = nullptr;

    int info = 0;

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void* d_work = nullptr;              /* device workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace for */

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(T) * N * N));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_W), sizeof(T) * N));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, &H_mat[0], sizeof(T) * N * N, hipMemcpyHostToDevice,
        stream));


    // step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(cusolverDnXsyevd_bufferSize(
        cusolverH, params, jobz, uplo, m, traits<T>::cuda_data_type, d_A, lda,
        traits<T>::cuda_data_type, d_W, traits<T>::cuda_data_type, &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice));

    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    // step 4: compute spectrum
    CUSOLVER_CHECK(cusolverDnXsyevd(
        cusolverH, params, jobz, uplo, m, traits<T>::cuda_data_type, d_A, lda,
        traits<T>::cuda_data_type, d_W, traits<T>::cuda_data_type, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info));

    CUDA_CHECK(hipMemcpyAsync(&V[0], d_A, sizeof(T) * N * N, hipMemcpyDeviceToHost,
        stream));
    CUDA_CHECK(hipMemcpyAsync(&W[0], d_W, sizeof(T) * N, hipMemcpyDeviceToHost,
        stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xsyevd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
    //CUDA_CHECK(hipDeviceReset());

}
template void SymmDiag_cuSOLVER_wrapper<float>(float* __restrict H_mat, float* __restrict W, float* __restrict V, const int N);